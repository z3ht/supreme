#include "hip/hip_runtime.h"
#include <iostream>
#include <SDL.h>
#include <hip/hip_runtime.h>


__device__ Uint32 getColor(int iter) {
    int r = (iter % 256);
    int g = ((iter * 2) % 256);
    int b = ((iter * 3) % 256);
    return (0xFF << 24) | (r << 16) | (g << 8) | b;
}

__global__ void mandelbrotSetKernel(unsigned int* output, float lowerX, float lowerY, float stepX, float stepY, int maxIter, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx >= width || idy >= height) return;
    int pixelIndex = (idy * width) + idx;

    float x0 = lowerX + idx * stepX;
    float y0 = lowerY + idy * stepY;
    float x = 0.0;
    float y = 0.0;

    int iteration = 0;
    while (x*x + y*y <= (2*2) && iteration < maxIter) {
        float xtemp = x*x - y*y + x0;
        y = 2*x*y + y0;
        x = xtemp;
        iteration++;
    }

    output[pixelIndex] = getColor(iteration);
}


int main() {
    const int width = 1024, height = 768, maxIter = 1000000;
    float lowerX = -2.5, lowerY = -1.0, upperX = 1.0, upperY = 1.0;

    float centerX = -1.186592f;
    float centerY = -1.901211e-1f;

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "Could not initialize SDL: " << SDL_GetError() << std::endl;
        return -1;
    }

    SDL_Window* window = SDL_CreateWindow("Mandelbrot Set", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, width, height, SDL_WINDOW_SHOWN);
    SDL_Surface* surface = SDL_GetWindowSurface(window);

    bool quit = false;
    SDL_Event event;

    while (!quit) {
        float zoomFactor = 1;

        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                quit = true;
            } else if (event.type == SDL_MOUSEWHEEL) {
                zoomFactor = (event.wheel.y > 0) ? 0.9f : 1.1f;

            } else if (event.type == SDL_MOUSEBUTTONDOWN) {
                int mouseX, mouseY;
                SDL_GetMouseState(&mouseX, &mouseY);

                centerX = lowerX + (mouseX / (float)width) * (upperX - lowerX);
                centerY = lowerY + (mouseY / (float)height) * (upperY - lowerY);
            }
        }

        float rangeX = (upperX - lowerX) * zoomFactor;
        float rangeY = (upperY - lowerY) * zoomFactor;

        lowerX = centerX - rangeX / 2;
        upperX = centerX + rangeX / 2;
        lowerY = centerY - rangeY / 2;
        upperY = centerY + rangeY / 2;

        float stepX = (upperX - lowerX) / width;
        float stepY = (upperY - lowerY) / height;

        unsigned int* d_output;
        hipMalloc(&d_output, width * height * sizeof(unsigned int));
        dim3 blocks(16, 16);
        dim3 grid((width + blocks.x - 1) / blocks.x, (height + blocks.y - 1) / blocks.y);
        mandelbrotSetKernel<<<grid, blocks>>>(d_output, lowerX, lowerY, stepX, stepY, maxIter, width, height);

        unsigned int* pixels = new unsigned int[width * height];
        hipMemcpy(pixels, d_output, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

        SDL_FillRect(surface, NULL, SDL_MapRGB(surface->format, 0, 0, 0));

        #pragma omp parallel for schedule(dynamic)
        for (int i = 0; i < width * height; ++i) {
            ((Uint32*)surface->pixels)[i] = pixels[i];
        }

        SDL_UpdateWindowSurface(window);

        hipFree(d_output);
        delete[] pixels;
    }

    SDL_Quit();

    return 0;
}
